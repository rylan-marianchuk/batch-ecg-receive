#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <thrust/sort.h>
#include <cstdlib>
#include <hipfft/hipfft.h>

__device__ float curvelength(float * signal, size_t SIGNAL_SIZE){
    float CL = 0;
    for (int j = 0; j < SIGNAL_SIZE - 1; j++) {
        float x1 = signal[j];
        float x2 = signal[j+1];
        CL += std::sqrt(1.0f + (x2 - x1)*(x2 - x1));
    }
    return CL;
}

__device__ float histogram_entropy(float * signal, size_t SIGNAL_SIZE, int bins=40){

    thrust::sort(thrust::seq, signal, signal + SIGNAL_SIZE);
    float min = signal[0];
    float max = signal[SIGNAL_SIZE - 1];
    float sum = 0;

    float binSize = (max - min) / bins;
    float binCount = 0;
    for (size_t i = 0; i < SIGNAL_SIZE; i++){
        if (signal[i] <= min + binSize){
            binCount++;
        }
        else {
            double v = binCount / binSize / SIGNAL_SIZE;
            sum += std::log2(v) * (v);
            binCount = 1;
            min += binSize;
        }
    }
    float v = binCount / binSize / SIGNAL_SIZE;
    sum += std::log2(v) * (v);

    return -sum;
}


__device__ float segment_autocorr_sim(float * signal, size_t SIGNAL_SIZE, int seg_size=1250, int nlags=50){
    // Demean
    float sum = 0.0f;
    for (size_t i = 0; i < seg_size; i++) sum += signal[i];
    float mean = sum / seg_size;

    for (size_t i = 0; i < seg_size; i++) signal[i] -= mean;

    int n = 2560;
    return 0.0f;
}


__device__ int has_flat20_samples(float * signal, size_t SIGNAL_SIZE){
    float prev = signal[0];
    short int longest = 0;
    for (size_t i = 1; i < SIGNAL_SIZE; i++){
        if (signal[i] == prev)
            longest++;
        else
            longest = 0;

        prev = signal[i];

        if (longest == 20)
            return 1;
    }
    return 0;
}

__global__ void GpuCompute(float * d_ecg_container,
                           float * d_resCL,  // Curve Length
                           float * d_resHE,  // Histogram entropy
                           float * d_resAC,  // Segment autocorrelation similarity
                           int * d_res20flat,  // 0 or 1 whether there is a flat line of length 20 (at least)
                           const size_t SIGNALS,
                           const size_t SIGNAL_SIZE){

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    while (i < SIGNALS){
        float * signal = d_ecg_container + (i * SIGNAL_SIZE);

        // Curve Length
        d_resCL[i] = curvelength(signal, SIGNAL_SIZE);

        // Histogram Entropy
        d_resHE[i] = histogram_entropy(signal, SIGNAL_SIZE);

        // Autocorrelation similarity
        d_resAC[i] = segment_autocorr_sim(signal, SIGNAL_SIZE);

        // 20 values equal in sequence?
        d_res20flat[i] = has_flat20_samples(signal, SIGNAL_SIZE);
        i += stride;
    }

}


extern "C" {
void GetWvfmFeaturesGPU(float * ecg_container,
                        float * resCL,  // Curve Length
                        float * resHE,  // Histogram entropy
                        float * resAC,  // Segment autocorrelation similarity
                        int * res20flat,  // 0 or 1 whether there is a flat line of length 20 (at least)
                        const size_t SIGNALS // number of signals in the 1D container
){
    hipError_t err = hipDeviceSetLimit(hipLimitMallocHeapSize, 1048576ULL*1024);
    // Declaring device pointers and initializing their memory
    float *d_ecg_container, *d_resCL, *d_resHE, *d_resAC;
    int * d_res20flat;

    // Defining the boundaries of the signals
    const size_t SIGNAL_SIZE = 5000;

    hipMalloc(&d_resCL, sizeof(float) * SIGNALS);
    hipMalloc(&d_resHE, sizeof(float) * SIGNALS);
    hipMalloc(&d_resAC, sizeof(float) * SIGNALS);
    hipMalloc(&d_res20flat, sizeof(int) * SIGNALS);

    hipMalloc(&d_ecg_container, sizeof(float) * SIGNALS * SIGNAL_SIZE);
    hipMemcpy(d_ecg_container, ecg_container, sizeof(float) * SIGNALS * SIGNAL_SIZE, hipMemcpyHostToDevice);

    const unsigned tpb_x = 256;
    const unsigned bpg_x = (SIGNALS + tpb_x - 1) / tpb_x;
    dim3 blocksPerGrid(bpg_x, 1, 1);
    dim3 threadsPerBlock(tpb_x, 1, 1);
    GpuCompute<<<blocksPerGrid, threadsPerBlock>>>(d_ecg_container, d_resCL, d_resHE, d_resAC, d_res20flat, SIGNALS, SIGNAL_SIZE);

    hipMemcpy(resCL, d_resCL, sizeof(float)*SIGNALS, hipMemcpyDeviceToHost);
    hipMemcpy(resHE, d_resHE, sizeof(float)*SIGNALS, hipMemcpyDeviceToHost);
    hipMemcpy(resAC, d_resAC, sizeof(float)*SIGNALS, hipMemcpyDeviceToHost);
    hipMemcpy(res20flat, d_res20flat, sizeof(int)*SIGNALS, hipMemcpyDeviceToHost);

    hipFree(d_ecg_container);
    hipFree(d_resCL);
    hipFree(d_resHE);
    hipFree(d_resAC);
    hipFree(d_res20flat);
}
}
