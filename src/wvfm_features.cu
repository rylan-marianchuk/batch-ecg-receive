#include "hip/hip_runtime.h"
//  Author:  Rylan Marianchuk
//  March 2022
//

#include <iostream>
#include <thrust/sort.h>
#include <cstdlib>


/*
    Return the curve length of the signal

    float * signal:     starting memory address of signal to compute
    size_t SIGNAL_SIZE: number of samples of the signal
*/
__device__ float curvelength(float * signal, size_t SIGNAL_SIZE){
    float CL = 0;
    for (int j = 0; j < SIGNAL_SIZE - 1; j++) {
        float x1 = signal[j];
        float x2 = signal[j+1];
        CL += std::sqrt(1.0f + (x2 - x1)*(x2 - x1));
    }
    return CL;
}


/*
    Return the entropy of the normalized histogram of the signal

    float * signal:     starting memory address of signal to compute
    size_t SIGNAL_SIZE: number of samples of the signal
    int bins:           bin amount of the histogram
*/
__device__ float histogram_entropy(float * signal, size_t SIGNAL_SIZE, int bins=40){

    thrust::sort(thrust::seq, signal, signal + SIGNAL_SIZE);
    float min = signal[0];
    float max = signal[SIGNAL_SIZE - 1];
    float sum = 0;

    float binSize = (max - min) / bins;
    float binCount = 0;
    for (size_t i = 0; i < SIGNAL_SIZE; i++){
        if (signal[i] <= min + binSize){
            binCount++;
        }
        else {
            double v = binCount / binSize / SIGNAL_SIZE;
            sum += std::log2(v) * (v);
            binCount = 1;
            min += binSize;
        }
    }
    float v = binCount / binSize / SIGNAL_SIZE;
    sum += std::log2(v) * (v);

    return -sum;
}


/*
    Return yes if the signal does not change amplitude for 20 samples

    float * signal:     starting memory address of signal to compute
    size_t SIGNAL_SIZE: number of samples of the signal
*/
__device__ int has_flat20_samples(float * signal, size_t SIGNAL_SIZE){
    float prev = signal[0];
    int longest = 0;
    for (size_t i = 1; i < SIGNAL_SIZE; i++){
        if (signal[i] == prev){
            longest++;
        }
        else{
            longest = 0;
        }

        prev = signal[i];

        if (longest == 20)
            return 1;
    }
    return 0;
}


/*
    CUDA Kernel - analogous parameters as below, but pointers are on device
    const size_t SIGNAL_SIZE:   the boundary of signals within the container
*/
__global__ void GpuCompute(float * d_ecg_container,
                           float * d_resCL,
                           float * d_resHE,
                           float * d_resAC,
                           int * d_res20flat,
                           const size_t SIGNALS,
                           const size_t SIGNAL_SIZE){

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    while (i < SIGNALS){
        float * signal = d_ecg_container + (i * SIGNAL_SIZE);

        // Curve Length
        d_resCL[i] = curvelength(signal, SIGNAL_SIZE);

        // Autocorrelation similarity
        //d_resAC[i] = segment_autocorr_sim(signal, SIGNAL_SIZE);

        // 20 values equal in sequence?
        d_res20flat[i] = has_flat20_samples(signal, SIGNAL_SIZE);

        // Histogram Entropy - do this last because of inplace sort
        d_resHE[i] = histogram_entropy(signal, SIGNAL_SIZE);
        i += stride;
    }

}


/*
    To invoke from python
    Input:
    float * ecg_container:    memory of the signals to compute features of

    To populate:
    float * resCL:            resCL[i] is the Curve Length of the ith signal given
    float * resHE:            resHE[i] is the Histogram Entropy of the ith signal given
    float * resAC:            not implemented
    int * res20flat:          binary vector, if ith value is 1, ith signal has some segment with no amplitude change for 20 samples

    const size_t SIGNALS:       number of signals within the container
*/
extern "C" {
void GetWvfmFeaturesGPU(float * ecg_container,
                        float * resCL,
                        float * resHE,
                        float * resAC,
                        int * res20flat,
                        const size_t SIGNALS
){
    // Increase the limit of the heap on device
    hipError_t err = hipDeviceSetLimit(hipLimitMallocHeapSize, 1048576ULL*1024);

    // Declaring device pointers and initializing their memory
    float *d_ecg_container, *d_resCL, *d_resHE, *d_resAC;
    int * d_res20flat;

    // Defining the boundaries of the signals
    const size_t SIGNAL_SIZE = 5000;

    // Allocate memory of result containers on the device
    hipMalloc(&d_resCL, sizeof(float) * SIGNALS);
    hipMalloc(&d_resHE, sizeof(float) * SIGNALS);
    hipMalloc(&d_resAC, sizeof(float) * SIGNALS);
    hipMalloc(&d_res20flat, sizeof(int) * SIGNALS);

    // Copy the input signals to device
    hipMalloc(&d_ecg_container, sizeof(float) * SIGNALS * SIGNAL_SIZE);
    hipMemcpy(d_ecg_container, ecg_container, sizeof(float) * SIGNALS * SIGNAL_SIZE, hipMemcpyHostToDevice);


    // Invoke Kernel
    const unsigned tpb_x = 256;
    const unsigned bpg_x = (SIGNALS + tpb_x - 1) / tpb_x;
    dim3 blocksPerGrid(bpg_x, 1, 1);
    dim3 threadsPerBlock(tpb_x, 1, 1);
    GpuCompute<<<blocksPerGrid, threadsPerBlock>>>(d_ecg_container, d_resCL, d_resHE, d_resAC, d_res20flat, SIGNALS, SIGNAL_SIZE);

    // Transfer result containers back to device
    hipMemcpy(resCL, d_resCL, sizeof(float)*SIGNALS, hipMemcpyDeviceToHost);
    hipMemcpy(resHE, d_resHE, sizeof(float)*SIGNALS, hipMemcpyDeviceToHost);
    hipMemcpy(resAC, d_resAC, sizeof(float)*SIGNALS, hipMemcpyDeviceToHost);
    hipMemcpy(res20flat, d_res20flat, sizeof(int)*SIGNALS, hipMemcpyDeviceToHost);

    hipFree(d_ecg_container);
    hipFree(d_resCL);
    hipFree(d_resHE);
    hipFree(d_resAC);
    hipFree(d_res20flat);
}
}
